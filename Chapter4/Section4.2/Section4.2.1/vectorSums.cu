
#include <hip/hip_runtime.h>
#include <iostream>

#define N 10

__global__ void add(int *a, int *b, int *c);

int main(void) {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;
    
    // allocate the memory on the GPU
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    // fill the arrays 'a' and 'b' on the CPU
    for (int i = 0; i < N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);
   
    // launch kernel
    add<<<N, 1>>>(dev_a, dev_b, dev_c);

    // copy the array 'c' back from the GPU to the CPU
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // display the results
    for (int i = 0; i < N; i++)
        std::cout << a[i] << " + " << b[i] << " = " << c[i] << std::endl;

    // free the memory allocated on the GPU
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}


__global__ void add(int *a, int *b, int *c) {
    int tid = blockIdx.x;    // handle the data at this index
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}
